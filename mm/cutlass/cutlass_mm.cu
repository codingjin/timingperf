#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/host/tensor_fill.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/host/tensor_copy.h>
#include <cutlass/util/tensor_view_io.h>

constexpr int ALIGN = 8;

int align(int n) {
    return (n + ALIGN - 1) / ALIGN * ALIGN;
}

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            std::cerr << "CUDA error " << err_ << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// CUTLASS GEMM configuration
using ElementInputA = cutlass::half_t;              // Input A data type (half-precision)
using ElementInputB = cutlass::half_t;              // Input B data type (half-precision)
using ElementOutput = float;                       // Output data type (single-precision)
using ElementAccumulator = float;                  // Accumulator data type (single-precision)
using LayoutInputA = cutlass::layout::ColumnMajor;  // Layout for A
using LayoutInputB = cutlass::layout::ColumnMajor;  // Layout for B
using LayoutOutput = cutlass::layout::ColumnMajor;  // Layout for C/D

// Tensor Core (TensorOp) configuration
using MMAOp = cutlass::arch::OpClassTensorOp;       // Use Tensor Cores
using SmArch = cutlass::arch::Sm80;                // Target architecture: SM80 (Ampere)

// Threadblock tile size
using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<256, 128, 32>;  // Threadblock tile size
using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 32>;           // Warp tile size
using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 8>;               // MMA operation tile size

// Epilogue operation
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                     // Data type of output matrix
    128 / cutlass::sizeof_bits<ElementOutput>::value,  // Number of elements per vectorized memory access
    ElementAccumulator,                                // Accumulator data type
    ElementAccumulator>;                               // Data type for alpha/beta

// Threadblock swizzle
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>;

// Number of pipeline stages
constexpr int NumStages = 2;

// Instantiate the CUTLASS GEMM kernel
using Gemm = cutlass::gemm::device::Gemm<
    ElementInputA, LayoutInputA,
    ElementInputB, LayoutInputB,
    ElementOutput, LayoutOutput,
    ElementAccumulator,
    MMAOp,
    SmArch,
    ShapeMMAThreadBlock,
    ShapeMMAWarp,
    ShapeMMAOp,
    EpilogueOp,
    SwizzleThreadBlock,
    NumStages>;

float get_median(float array[], int size) {
    std::vector<float> temp(array, array + size);
    std::sort(temp.begin(), temp.end());
    if (size % 2 == 1) {
        return temp[size / 2];
    } else {
        return (temp[size / 2 - 1] + temp[size / 2]) / 2.0f;
    }
}

void fill_random(std::vector<cutlass::half_t>& data, int size, float range = 1.0f) {
    for (int i = 0; i < size; ++i) {
        data[i] = static_cast<cutlass::half_t>(static_cast<float>(rand()) / RAND_MAX * range);
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cerr << "Usage: ./cutlass_mm <M> <N> <K>" << std::endl;
        return EXIT_FAILURE;
    }

    const int M = align(atoi(argv[1]));
    const int N = align(atoi(argv[2]));
    const int K = align(atoi(argv[3]));

    std::cout << "M=" << M << " N=" << N << " K=" << K << std::endl;

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Problem size
    cutlass::gemm::GemmCoord problem_size(M, N, K);

    // Host memory allocation
    std::vector<cutlass::half_t> host_A(M * K);
    std::vector<cutlass::half_t> host_B(K * N);
    std::vector<float> host_C(M * N);

    // Fill host memory with random data
    fill_random(host_A, host_A.size());
    fill_random(host_B, host_B.size());
    std::fill(host_C.begin(), host_C.end(), 0.0f);

    // Device memory allocation
    cutlass::half_t* A;
    cutlass::half_t* B;
    float* C;
    CUDA_CHECK(hipMalloc(&A, host_A.size() * sizeof(cutlass::half_t)));
    CUDA_CHECK(hipMalloc(&B, host_B.size() * sizeof(cutlass::half_t)));
    CUDA_CHECK(hipMalloc(&C, host_C.size() * sizeof(float)));

    // Copy data from host to device
    CUDA_CHECK(hipMemcpy(A, host_A.data(), host_A.size() * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(B, host_B.data(), host_B.size() * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(C, host_C.data(), host_C.size() * sizeof(float), hipMemcpyHostToDevice));

    // Instantiate CUTLASS GEMM
    Gemm gemm_op;

    // Create arguments for GEMM
    typename Gemm::Arguments arguments{
        problem_size,
        {A, M},
        {B, K},
        {C, M},
        {C, M},
        {alpha, beta}
    };

    // Check if the problem size is supported
    cutlass::Status status = gemm_op.can_implement(arguments);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS GEMM configuration not supported" << std::endl;
        return EXIT_FAILURE;
    }

    // Allocate workspace
    size_t workspace_size = gemm_op.get_workspace_size(arguments);
    void* workspace = nullptr;
    if (workspace_size > 0) {
        CUDA_CHECK(hipMalloc(&workspace, workspace_size));
    }

    // Warm-up
    for (int i = 0; i < 3; ++i) {
        status = gemm_op(arguments, workspace);
        CUDA_CHECK(hipDeviceSynchronize());
        if (status != cutlass::Status::kSuccess) {
            std::cerr << "CUTLASS GEMM failed during warm-up" << std::endl;
            return EXIT_FAILURE;
        }
    }

    // Timing
    float time_array[9];
    hipEvent_t start, stop;
    for (int i = 0; i < 9; ++i) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        status = gemm_op(arguments, workspace);
        CUDA_CHECK(hipDeviceSynchronize());

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time_array[i], start, stop);

        hipEventDestroy(start);
        hipEventDestroy(stop);

        if (status != cutlass::Status::kSuccess) {
            std::cerr << "CUTLASS GEMM failed during timing" << std::endl;
            return EXIT_FAILURE;
        }
    }

    // Calculate median time
    float median_time = get_median(time_array, 9);
    std::cout << "Median time: " << median_time << " ms" << std::endl;

    // Calculate performance
    float gflops = 2.0f * M * N * K / (median_time * 1.0e6f);
    std::cout << "Performance: " << gflops << " GFLOPs" << std::endl;

    // Free memory
    CUDA_CHECK(hipFree(A));
    CUDA_CHECK(hipFree(B));
    CUDA_CHECK(hipFree(C));
    if (workspace) {
        CUDA_CHECK(hipFree(workspace));
    }

    return 0;
}